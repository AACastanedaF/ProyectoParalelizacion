//Proyecto final en cuda
//Alumno: Angel Adrian Castañeda Flores
//El codigo se encuentra mejor explicado en el reporte
//Librerias a utilizar
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>
//Suma de matrices
__global__ void Suma(int* a, int* b, int* c, int m, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < m && j < n) {
        c[(n * i) + j] = a[(n * i) + j] + b[(n * i) + j];
    }
}
//Resta de matrices
__global__ void Resta(int* a, int* b, int* c, int m, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < m && j < n) {
        c[(n * i) + j] = a[(n * i) + j] - b[(n * i) + j];
    }
}
//Tranpuesta de Matriz
__global__ void Transpuesta(int* a, int* c, int m, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < m && j < n) {
        c[(n * i) + j] = a[(m * j) + i];
    }
}
//Mutliplicacion de Matrices
__global__ void Matmul(int* a, int* b, int* c, int m, int n, int ca, int cb) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int aux = 0;
    if (i < m && j < n) {
        for (int k = 0; k < ca; k++) {
            aux = (a[(ca * i) + k] * b[(cb * k) + j]) + aux;
        }
        c[(n * i) + j] = aux;
    }
}
using namespace std;
int main() {
    //Semilla para los numeros aleatorios
    srand(time(NULL));
    //matrices y propiedades
    int *matrizA, *matrizB, *matrizC;
    int *cudaA, *cudaB, *cudaC;
    int filasA, filasB, filasC;
    int columnasA, columnasB, columnasC;
    int aux;
    //variables para medir tiempo
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    filasA = 10;
    columnasA = 10;
    filasB = 10;
    columnasB = 10;
    //Asigacion de memoria a la matriz A
    matrizA = new int[filasA * columnasA];
    matrizB = new int[filasB * columnasB];
    //Generacion de matriz A
    for (int i = 0; i < filasA; i++) {
        for (int j = 0; j < columnasA; j++) {
            matrizA[(columnasA * i) + j] = (rand() % 10+1);
        }
    }
    //Impresion de matriz A
    cout << "Matriz A" << endl;
    for (int i = 0; i < filasA; i++) {
        for (int j = 0; j < columnasA; j++) {
            cout << matrizA[(columnasA * i) + j] << "\t";
        }
        cout << endl;
    }
    //Generacion de matriz B
    for (int i = 0; i < filasB; i++) {
        for (int j = 0; j < columnasB; j++) {
            matrizB[(columnasB * i) + j] = (rand() % 10+1);
        }
    }
    //Impresion de matriz B
    cout << "Matriz B" << endl;
    for (int i = 0; i < filasB; i++) {
        for (int j = 0; j < columnasB; j++) {
            cout << matrizB[(columnasB * i) + j] << "\t";
        }
        cout << endl;
    }
    //Asignacion de memoria en el GPU para las matrices A y B
    hipMalloc((void**)&cudaA, filasA * columnasA * sizeof(int));
    hipMemcpy(cudaA, matrizA, filasA * columnasA * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**)&cudaB, filasB * columnasB * sizeof(int));
    hipMemcpy(cudaB, matrizB, filasB * columnasB * sizeof(int), hipMemcpyHostToDevice);
    //Suma de Matrices A con B
    if (filasA == filasB && columnasA == columnasB){
        hipEventRecord(start);
        //Se inicializa C
        filasC = filasA;
        columnasC = columnasA;
        matrizC = new int[filasC * columnasC];
        //Se asigna memoria a la GPU para C
        hipMalloc((void**)&cudaC, filasC * columnasC * sizeof(int));
        //Generacion de la malla
        dim3 threadsPerBlock(8,8);
        dim3 numBlocks(ceil(float(filasC) / float(threadsPerBlock.x)), ceil(float(columnasC) / float(threadsPerBlock.y)));
        //Se ejecuta la suma en el GPU
        Suma <<<numBlocks, threadsPerBlock>>> (cudaA, cudaB, cudaC, filasC, columnasC);
        //Esperar a que el GPU termine
        hipDeviceSynchronize();
        //Se copia el resultado del GPU al CPU
        hipMemcpy(matrizC, cudaC, filasC * columnasC * sizeof(int), hipMemcpyDeviceToHost);
        //Impresion de la suma
        cout << "La suma de A con B es: " << endl;
        for (int i = 0; i < filasC; i++) {
            for (int j = 0; j < columnasC; j++) {
                cout << matrizC[(columnasC * i) + j] << "\t";
            }
            cout << endl;
        }
        //Liberacion de Memoria de C
        hipFree(cudaC);
        delete[] matrizC;
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        cout << "el tiempo de ejecución de la suma es de: " << milliseconds/1000 << endl;
    } 
    else {
        cout << "No se puede hacer la suma de matrices por incompatibilidad de dimensiones" << endl;
    }   
    //Resta de Matrices A con B
    if (filasA == filasB && columnasA == columnasB) {
        hipEventRecord(start);
        //Se inicializa C
        filasC = filasA;
        columnasC = columnasA;
        matrizC = new int[filasC * columnasC];
        //Se asigna memoria a la GPU para C
        hipMalloc((void**)&cudaC, filasC * columnasC * sizeof(int));
        hipMemcpy(cudaC, matrizC, filasC * columnasC * sizeof(int), hipMemcpyHostToDevice);
        //Generacion de la malla
        dim3 threadsPerBlock(8, 8);
        dim3 numBlocks(ceil(float(filasC) / float(threadsPerBlock.x)), ceil(float(columnasC) / float(threadsPerBlock.y)));
        //Se ejecuta la suma en el GPU
        Resta <<<numBlocks, threadsPerBlock>>> (cudaA, cudaB, cudaC,filasC, columnasC);
        //Se copia el resultado del GPU al CPU
        hipMemcpy(matrizC, cudaC, filasC * columnasC * sizeof(int), hipMemcpyDeviceToHost);
        //Impresion de la resta de A con B
        cout << "La resta de A con B es: " << endl;
        for (int i = 0; i < filasC; i++) {
            for (int j = 0; j < columnasC; j++) {
                cout << matrizC[(columnasC * i) + j] << "\t";
            }
            cout << endl;
        }
        //Liberacion de Memoria de C
        hipFree(cudaC);
        delete[] matrizC;
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        cout << "el tiempo de ejecución de la resta es de: " << milliseconds / 1000 << endl;
    }
    else {
        cout << "No se puede hacer la resta de matrices por incompatibilidad de dimensiones" << endl;
    }
    //Transpuesta de A
    //Se inicializa C
    hipEventRecord(start);
    filasC = columnasA;
    columnasC = filasA;
    matrizC = new int[filasC * columnasC];
    //Se asigna memoria a la GPU para C
    hipMalloc((void**)&cudaC, filasC * columnasC * sizeof(int));
    hipMemcpy(cudaC, matrizC, filasC * columnasC * sizeof(int), hipMemcpyHostToDevice);
    //Generacion de la malla
    dim3 threadsPerBlock(8, 8);
    dim3 numBlocks(ceil(float(filasC) / float(threadsPerBlock.x)), ceil(float(columnasC) / float(threadsPerBlock.y)));
    //Se ejecuta la transpuesta en el GPU
    Transpuesta <<<numBlocks, threadsPerBlock >>> (cudaA, cudaC, filasC, columnasC);
    //Se copia el resultado del GPU al CPU
    hipMemcpy(matrizC, cudaC, filasC * columnasC * sizeof(int), hipMemcpyDeviceToHost);
    //Impresion de transpuesta de A con B
    cout << "La transpuesta de A: " << endl;
    for (int i = 0; i < filasC; i++) {
        for (int j = 0; j < columnasC; j++) {
            cout << matrizC[(columnasC * i) + j] << "\t";
        }
        cout << endl;
    }
    //Liberacion de Memoria de C
    hipFree(cudaC);
    delete[] matrizC;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "el tiempo de ejecución de la transpuesta es de: " << milliseconds / 1000 << endl;
    //Multiplicacion de Matrices A con B
    if (columnasA == filasB) {
        hipEventRecord(start);
        //Se inicializa C
        filasC = filasA;
        columnasC = columnasB;
        matrizC = new int[filasC * columnasC];
        //Se asigna memoria a la GPU para C
        hipMalloc((void**)&cudaC, filasC * columnasC * sizeof(int));
        hipMemcpy(cudaC, matrizC, filasC * columnasC * sizeof(int), hipMemcpyHostToDevice);
        //Generacion de la malla
        dim3 threadsPerBlock(8, 8);
        dim3 numBlocks(ceil(float(filasC) / float(threadsPerBlock.x)), ceil(float(columnasC) / float(threadsPerBlock.y)));
        //Se ejecuta la suma en el GPU
        Matmul <<<numBlocks, threadsPerBlock>>> (cudaA, cudaB, cudaC, filasC, columnasC, columnasA, columnasB);
        //Se copia el resultado del GPU al CPU
        hipMemcpy(matrizC, cudaC, filasC * columnasC * sizeof(int), hipMemcpyDeviceToHost);
        //Impresion de la resta de A con B
        cout << "La multiplicacion de A con B es: " << endl;
        for (int i = 0; i < filasC; i++) {
            for (int j = 0; j < columnasC; j++) {
                cout << matrizC[(columnasC * i) + j] << "\t";
            }
           cout << endl;
        }
        //Liberacion de Memoria de C
        hipFree(cudaC);
        delete[] matrizC;
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        cout << "el tiempo de ejecución de la multiplicacion es de: " << milliseconds / 1000 << endl;
    }
    else {
        cout << "No se puede hacer la multiplicacion de matrices por incompatibilidad de dimensiones" << endl;
    }
    //liberacion de memoria
    hipFree(cudaA);
    delete[] matrizA;
    hipFree(cudaB);
    delete[] matrizB;
    return 0;  
}